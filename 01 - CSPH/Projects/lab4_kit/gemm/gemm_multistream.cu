#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>
#include <vector>

#include "gemm_singleFP32.cuh"


#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans)                \
  {                                        \
    cudaAssert((ans), __FILE__, __LINE__); \
  }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}
// cublas API error checking
#define CUBLAS_CHECK(err)                                                  \
  do {                                                                     \
    hipblasStatus_t err_ = (err);                                           \
    if (err_ != HIPBLAS_STATUS_SUCCESS) {                                   \
      std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__); \
      throw std::runtime_error("cublas error");                            \
    }                                                                      \
  } while (0)
#else
#define cudaCheckError(ans) ans
#define CUBLAS_CHECK(ans) ans
#endif

// Multi-stream custom FP32 kernel
// Here we will run one GEMM per stream using the cudaBlockKernel_FP32 from the previous lab. It will process "batch count" GEMMs, one per stream.
void gemm_fp32_multiStream(float **A_batch, float **B_batch, float **C_batch, int m, int n, int k, int batchCount, hipStream_t *streams, bool warmup = false) {
  
  // Declare device pointers arrays, each vector will store the pointers to the matrices in the GPU
  std::vector<float*> devA(batchCount), devB(batchCount), devC(batchCount);

  // Allocate memory and copy the matrices to the GPU, each operation is assigned to its stream. Notice the usage of Async
  for (int b = 0; b < batchCount; b++) {
    hipMallocAsync((void**)&devA[b], sizeof(float)*m*k, streams[b]); 
    hipMallocAsync((void**)&devB[b], sizeof(float)*k*n, streams[b]);
    hipMallocAsync((void**)&devC[b], sizeof(float)*m*n, streams[b]);
    hipMemcpyAsync(devA[b], A_batch[b], sizeof(float)*m*k, hipMemcpyHostToDevice, streams[b]);
    hipMemcpyAsync(devB[b], B_batch[b], sizeof(float)*k*n, hipMemcpyHostToDevice, streams[b]);
    hipMemcpyAsync(devC[b], C_batch[b], sizeof(float)*m*n, hipMemcpyHostToDevice, streams[b]);
  }
  // Wait for all streams to finish
  hipDeviceSynchronize();
  int N = n;
  dim3 threadsPerBlock(LBLK, LBLK);
  dim3 blocks(updiv(n, LBLK), updiv(n, LBLK));
    // Timing only the kernel launches, as done on the other profilings
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
 // Iterate by each matrix pair, assigning its GEMM to a Stream
  for(int b = 0; b < batchCount; b++) {
    //For example, here we could use only 4 streams and where b is, we would use b % 4. This would lead to a different work distribution. 
    //The 0 in the arguments means we don't use Shared Memory
    cudaBlockKernel_FP32<<<blocks, threadsPerBlock, 0, streams[b]>>>(N, devA[b], devB[b], devC[b]);
  }
  // We don't need to sync after the kernel, only after the memory transfer so that we synchronize all the streams. 
  for(int b = 0; b < batchCount; b++) {
    hipMemcpyAsync(C_batch[b], devC[b], sizeof(float)*m*n, hipMemcpyDeviceToHost, streams[b]);
  }
  hipDeviceSynchronize(); // this will synchronize all the streams

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  if (!warmup) {
    printf("Multi-stream FP32 (kernel only): %d batches, Time: %.3f ms, GFLOPS: %.3f\n",
           batchCount, elapsedTime, batchCount * 2.0 * m * n * k / (elapsedTime * 1e-3) / 1e9);
  }
  // Free the memory of each matrix
  for (int b = 0; b < batchCount; b++) {
    hipFree(devA[b]);
    hipFree(devB[b]);
    hipFree(devC[b]);
  }
}

// Multi-stream cuBLAS GEMM (FP32/FP16/TF32)
// the logic for copying and allocating the data is similar to the gemm_fp32_multiStream version
// To use multiple streams here, you need to bind the stream to the calling handle, the stream is already created. So, it would be like
// hipblasSetStream(handle, streams[i]);
// More information at https://docs.nvidia.com/cuda/cublas/#parallelism-with-streams

void cublas_gemm_multiStream(
    float **A_batch, float **B_batch, float **C_batch,
    int m, int n, int k, int batchCount,
    hipStream_t *streams,
    const char* mode,
    hipblasComputeType_t computeType,
    bool warmup = false
) {
    // Declare device pointers arrays, each vector will store the pointers to the matrices in the GPU
    std::vector<float*> d_A(batchCount), d_B(batchCount), d_C(batchCount);

    // Create CuBLAS handle as usual 
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Allocate memory and copy the matrices to the GPU, each operation is assigned to its stream. Notice the usage of Async
    for (int i = 0; i < batchCount; i++) {
        hipMallocAsync(&d_A[i], sizeof(float)*m*k, streams[i]);
        hipMallocAsync(&d_B[i], sizeof(float)*k*n, streams[i]);
        hipMallocAsync(&d_C[i], sizeof(float)*m*n, streams[i]);

        hipMemcpyAsync(d_A[i], A_batch[i], sizeof(float)*m*k, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_B[i], B_batch[i], sizeof(float)*k*n, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_C[i], C_batch[i], sizeof(float)*m*n, hipMemcpyHostToDevice, streams[i]);
    }
    // Wait for all streams to finish
    hipDeviceSynchronize();

    const float alpha = 1.f, beta = 0.f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    if (!warmup) hipEventRecord(start, 0);

     // Iterate by each matrix pair, assigning its GEMM to a Stream
    for (int i = 0; i < batchCount; i++) {
        CUBLAS_CHECK(hipblasSetStream(handle, streams[i])); // bind the handle to a stream
        // Here the parameters are the same as in our single stream version
        CUBLAS_CHECK(hipblasGemmEx(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            m, n, k,
            &alpha,
            d_A[i], HIP_R_32F, m,
            d_B[i], HIP_R_32F, k,
            &beta,
            d_C[i], HIP_R_32F, m,
            computeType, HIPBLAS_GEMM_DEFAULT
        ));
    }
    
    hipDeviceSynchronize();

    if (!warmup) {
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("Multi-stream cuBLAS (%s, %s, kernel only): %d batches, Time: %.3f ms, GFLOPS: %.3f\n",
               mode, cublasComputeTypeToString(computeType), batchCount, elapsedTime,
               batchCount * 2.0 * m * n * k / (elapsedTime * 1e-3) / 1e9);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy results back asynchronously
    for (int i = 0; i < batchCount; i++) {
        hipMemcpyAsync(C_batch[i], d_C[i], sizeof(float)*m*n, hipMemcpyDeviceToHost, streams[i]);
    }
    hipDeviceSynchronize(); // synchronize all the streams
    // free the memory
    for (int i = 0; i < batchCount; i++) {
        hipFree(d_A[i]);
        hipFree(d_B[i]);
        hipFree(d_C[i]);
    }
    CUBLAS_CHECK(hipblasDestroy(handle));
}


