#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>
#include <vector>

#include "gemm_singleFP32.cuh"

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans)                \
  {                                        \
    cudaAssert((ans), __FILE__, __LINE__); \
  }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}
// cublas API error checking
#define CUBLAS_CHECK(err)                                                  \
  do {                                                                     \
    hipblasStatus_t err_ = (err);                                           \
    if (err_ != HIPBLAS_STATUS_SUCCESS) {                                   \
      std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__); \
      throw std::runtime_error("cublas error");                            \
    }                                                                      \
  } while (0)
#else
#define cudaCheckError(ans) ans
#define CUBLAS_CHECK(ans) ans
#endif

// This is the batched version of our typical CuBLAS kernel. Instead of a single GEMM, it will perform as many as we tell it to. In this case, it is given by the argument "batch count".
// In other words, it means that we will command the kernel to perform "batch count" AxB matrix multiplications.
// It only uses a single stream and computes all the results in a single kernel. The logic is the same as the previous functions but with some extra details. This will be explained in the comments near the code.
// For more information about hipblasGemmBatchedEx, visit: https://docs.nvidia.com/cuda/cublas/#cublasgemmbatchedex
void cublas_gemm_batched_fp32(
    float **A_batch, float **B_batch, float **C_batch,
    int m, int n, int k, int batchCount,
    hipStream_t *streams,
    const char* mode,
    hipblasComputeType_t computeType,
    bool warmup = false
) {

    //Instead of a single matrix like in the previous GEMMs, we will be feeding the CuBLAS function "batch count" GEMMs. For this we need pointers for each matrix, both for CPU and GPU matrices
    //Here we allocate CPU pointers
    std::vector<float*> d_A(batchCount), d_B(batchCount), d_C(batchCount); //here we are allocating an array of float pointers: Each pointer will point to a matrix
    std::vector<float*> h_Aptrs(batchCount), h_Bptrs(batchCount), h_Cptrs(batchCount); //here we are allocating an array of float pointers: Each pointer will point to a matrix

    // Here we allocate memory and copy each pair of matrices to the GPU. 
    for (int i = 0; i < batchCount; i++) {
      hipMalloc(&d_A[i], sizeof(float)*m*k);
      hipMalloc(&d_B[i], sizeof(float)*k*n);
      hipMalloc(&d_C[i], sizeof(float)*m*n);
      hipMemcpy(d_A[i], A_batch[i], sizeof(float)*m*k, hipMemcpyHostToDevice);
      hipMemcpy(d_B[i], B_batch[i], sizeof(float)*k*n, hipMemcpyHostToDevice);
      hipMemcpy(d_C[i], C_batch[i], sizeof(float)*m*n, hipMemcpyHostToDevice);
      //here we store the pointers to where the matrices are stored in the GPU. This is needed later as an argument of the hipblasGemmBatchedEx()
      h_Aptrs[i] = d_A[i]; 
      h_Bptrs[i] = d_B[i];
      h_Cptrs[i] = d_C[i];
    }
    // Here we simply allocate pointers of pointers to store where the matrices are stored in the memory. 
    float **d_Aptrs, **d_Bptrs, **d_Cptrs;
    hipMalloc(&d_Aptrs, batchCount * sizeof(float*));
    hipMalloc(&d_Bptrs, batchCount * sizeof(float*));
    hipMalloc(&d_Cptrs, batchCount * sizeof(float*));

    // we use .data() because std::vector<float*> has that built-in function to retrieve its pointer 
    hipMemcpy(d_Aptrs, h_Aptrs.data(), batchCount * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_Bptrs, h_Bptrs.data(), batchCount * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_Cptrs, h_Cptrs.data(), batchCount * sizeof(float*), hipMemcpyHostToDevice);

    // Create the cublas handle as usual
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    const float alpha = 1.f, beta = 0.f;

  /* DO NOT MODIFY THIS PART
   * This part of the code is responsible for accurately measuring the time taken by the kernel.
   * Here the time is recorded and printed.
   * The performance is calculated in GFLOPS.
   */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (!warmup) hipEventRecord(start, 0);

    CUBLAS_CHECK(hipblasGemmBatchedEx(
    handle,                 // cuBLAS context handle
    HIPBLAS_OP_N,            
    HIPBLAS_OP_N,           
    m,                    
    n,                    
    k,                     
    &alpha,                
    (void**)d_Aptrs,        // device array of pointers to A matrices
    HIP_R_32F,             // data type of A
    m,                     
    (void**)d_Bptrs,        // device array of pointers to B matrices
    HIP_R_32F,             // data type of B
    k,                      
    &beta,                  // scalar multiplier for existing C
    (void**)d_Cptrs,        // device array of pointers to C matrices
    HIP_R_32F,             // data type of C
    m,                      
    batchCount,             // number of matrices in the batch
    computeType,            // computation precision/type
    HIPBLAS_GEMM_DEFAULT     // algorithm selection (default)
));

    if (!warmup) {
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("Batched cuBLAS (%s, %s, kernel only): %d batches, Time: %.3f ms, GFLOPS: %.3f\n",
               mode, cublasComputeTypeToString(computeType), batchCount, elapsedTime,
               batchCount * 2.0 * m * n * k / (elapsedTime * 1e-3) / 1e9);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);


    // Copy the results back from the GPU
    for (int i = 0; i < batchCount; i++) {
      hipMemcpy(C_batch[i], d_C[i], sizeof(float)*m*n, hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    // Free the memory 
    for (int i = 0; i < batchCount; i++) {
      hipFree(d_A[i]);
      hipFree(d_B[i]);
      hipFree(d_C[i]);
    }
    hipFree(d_Aptrs);
    hipFree(d_Bptrs);
    hipFree(d_Cptrs);
    CUBLAS_CHECK(hipblasDestroy(handle));
}

